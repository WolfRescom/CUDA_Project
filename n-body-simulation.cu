#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cstdlib>
#include <cmath>

typedef struct{
    float2 pos;
    float2 vel;
    float2 acc;
    float mass;
} Body;

__global__ void update(Body* bodies, int n, float dt){
    for (int i = 0; i < n; i++) {
        for (int j = i + 1; j < n; j++) {
            float2 p1 = bodies[i].pos;
            float m1 = bodies[i].mass;
            float2 p2 = bodies[j].pos;
            float m2 = bodies[j].mass;

            float dx = bodies[j].pos.x - bodies[i].pos.x;
            float dy = bodies[j].pos.y - bodies[i].pos.y;
            float dist = sqrt(dx * dx + dy * dy);

            float2 r = make_float2(dx / dist, dy / dist);

            float grav = r / (dist * dist * dist);

            bodies[i].acc += m2 * grav;
            bodies[j].acc -= m1 * grav;
        }
    }
}

//Host Code
int main() {
    const int N = 1024; // number of bodies
    const float  dt     = 0.01f;
    const int    steps  = 1000;

    std::vector<Body> hostP(N);
    for(int i = 0; i < N; ++i){
        hostP[i].pos = make_float2(rand()/float(RAND_MAX), rand()/float(RAND_MAX));
        hostP[i].vel = make_float2(0.0f, 0.0f);
        hostP[i].acc = make_float2(0.0f, 0.0f);
        hostP[i].mass = 1.0f;
    }

    Body* devP;
    hipMalloc(&devP, N * sizeof(Body));
    hipMemcpy(devP, hostP.data(), N * sizeof(Body), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    for (int t = 0; t < steps; ++t) {
        update<<<blocks, threadsPerBlock>>>(devP, N, dt);
        hipDeviceSynchronize();
    }

    std::cout << "Body[0] pos = ("
        << hostP[0].pos.x << ", "
        << hostP[0].pos.y << ")\n";


    hipFree(devP);
    return 0;
}